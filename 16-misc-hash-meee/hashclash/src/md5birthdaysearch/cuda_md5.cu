#include "hip/hip_runtime.h"
/**************************************************************************\
|
|    Copyright (C) 2009 Marc Stevens
|
|    This program is free software: you can redistribute it and/or modify
|    it under the terms of the GNU General Public License as published by
|    the Free Software Foundation, either version 3 of the License, or
|    (at your option) any later version.
|
|    This program is distributed in the hope that it will be useful,
|    but WITHOUT ANY WARRANTY; without even the implied warranty of
|    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
|    GNU General Public License for more details.
|
|    You should have received a copy of the GNU General Public License
|    along with this program.  If not, see <http://www.gnu.org/licenses/>.
|
\**************************************************************************/

#include <iostream>
#include <vector>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <boost/cstdint.hpp>
#include "cuda_cyclicbuffer.hpp"

using namespace std;

typedef boost::uint32_t uint32;
typedef boost::uint64_t uint64;

#define MAX_CUDA_THREADS (1<<20)
#define MAX_CUDA_BLOCKS 256
#define MAX_CUDA_THREADS_PER_BLOCK 2048
#define REGISTERS_PER_CUDA_THREAD 64

#define TRAIL_NOCONSTRUCTOR
#include "birthday_types.hpp"

#ifndef CUDA_SAFE_CALL
#define CUDA_SAFE_CALL(s) { auto ce = s; if (ce != hipSuccess) { throw std::runtime_error("CUDA API Error:\n" + std::string(hipGetErrorName(ce)) + ":\n" + std::string(hipGetErrorString(ce))); } }
#endif

#ifndef cutilSafeCall
#define cutilSafeCall(s) (s)
#endif


/****
  NOTE WARNING: 
  We assume that all global __device__ variables below are *thread* *specific*
 (instead of global) storage managed by the cuda realtime libraries 
*****/
// last template parameter is fence type: 0=none, 1=block, 2=gpu
typedef cyclic_buffer_cas_t<MAX_CUDA_THREADS,uint32,7,cyclic_buffer_control_cas_t<MAX_CUDA_THREADS>,2> state_buffer_t;
typedef cyclic_buffer_mask_t<MAX_CUDA_THREADS_PER_BLOCK,uint32,7,cyclic_buffer_control_mask_t<MAX_CUDA_THREADS_PER_BLOCK>,1> work_buffer_t;
typedef work_buffer_t::control_t work_control_t;
typedef cyclic_buffer_cas_t<MAX_CUDA_THREADS,uint32,15,cyclic_buffer_control_cas_t<MAX_CUDA_THREADS>,2> collisions_buffer_t;
typedef collisions_buffer_t::control_t collisions_control_t;

// static gpu buffer that always stays on GPU
__device__ state_buffer_t gworking_states;
__device__ collisions_buffer_t gcollision_states;

// per-block buffer for trails
__device__ work_buffer_t gtrailsout_buf[MAX_CUDA_BLOCKS];
__device__ work_control_t gtrailsout_ctl[MAX_CUDA_BLOCKS];
__shared__ work_control_t gtrailsout_ctlblock;

// gpu-wide in- and out-put buffers collisions
__device__ collisions_buffer_t gcollisionsin_buf;
__device__ collisions_buffer_t gcollisionsout_buf;
__device__ collisions_control_t gcollisionsin_ctl;
__device__ collisions_control_t gcollisionsout_ctl;
__device__ volatile uint32 halt_flag;

__constant__ uint32 msg1[16], msg2[16], ihv1[4], ihv2[4], ihv2mod[4];
__constant__ uint32 precomp1[4], precomp2[4];
__constant__ uint32 hybridmask, distinguishedpointmask, maximumpathlength;


class cuda_device_detail {
public:
	uint32 device;
	uint32 blocks;
	uint32 threadsperblock;
	work_buffer_t* trailsout_buf;
	work_control_t* trailsout_ctl;

	// host-side buffer
	size_t nrcollisions_on_gpu;
	vector< pair<trail_type,trail_type> > collisions;
	collisions_buffer_t* collisionsin_buf;
	collisions_control_t* collisionsin_ctl;
	collisions_buffer_t* collisionsout_buf;
	collisions_control_t* collisionsout_ctl;
};


/* F, G and H are basic MD5 functions: selection, majority, parity */
#define MD5_F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define MD5_G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define MD5_H(x, y, z) ((x) ^ (y) ^ (z))
#define MD5_I(x, y, z) ((y) ^ ((x) | (~z)))

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define MD5_FF(a, b, c, d, x, s, ac) \
  {(a) += MD5_F ((b), (c), (d)) + (x) + (uint32)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define MD5_GG(a, b, c, d, x, s, ac) \
  {(a) += MD5_G ((b), (c), (d)) + (x) + (uint32)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define MD5_HH(a, b, c, d, x, s, ac) \
  {(a) += MD5_H ((b), (c), (d)) + (x) + (uint32)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }
#define MD5_II(a, b, c, d, x, s, ac) \
  {(a) += MD5_I ((b), (c), (d)) + (x) + (uint32)(ac); \
   (a) = ROTATE_LEFT ((a), (s)); \
   (a) += (b); \
  }


__device__ void backup_controls()
{
	__syncthreads();
	if (threadIdx.x == 0)
	{
		gtrailsout_ctlblock = gtrailsout_ctl[blockIdx.x];
	}
	__syncthreads();
}

__device__ void restore_controls()
{
	__syncthreads();
	if (threadIdx.x == 0)
	{
		 gtrailsout_ctl[blockIdx.x] = gtrailsout_ctlblock;
	}
	__syncthreads();
}


__global__ void cuda_md5_init()
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	gworking_states.get_ref<6>(idx) = 0; // len = 0
	gcollision_states.get_ref<14>(idx) = 1; // bad = 1
	if (threadIdx.x == 0)
	{
		gtrailsout_buf[blockIdx.x].reset(gtrailsout_ctl[blockIdx.x]);
		gcollisionsin_buf.reset(gcollisionsin_ctl);
		gcollisionsout_buf.reset(gcollisionsout_ctl);
	}

}

bool cuda_device::init(uint32 device, const uint32 ihv1b[4], const uint32 ihv2b[4], const uint32 ihv2modb[4], const uint32 msg1b[16], const uint32 msg2b[16], uint32 hmask, uint32 dpmask, uint32 maxlen)
{
	detail = new cuda_device_detail;
	detail->device = device;

    int deviceCount;
    CUDA_SAFE_CALL( hipGetDeviceCount(&deviceCount) );
    if (deviceCount == 0) {
        cout << "There is no device supporting CUDA!" << endl;
        return false;
	}
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, device) );
    if (deviceProp.major == 9999) {
		cout << "Emulation device found." << endl;
		return false;
	}
	cout << "CUDA device " << device << ": " << deviceProp.name << " (" << deviceProp.multiProcessorCount << " MPs)" << endl;
	unsigned maxthreadspermp = deviceProp.maxThreadsPerMultiProcessor;
	if (maxthreadspermp > MAX_CUDA_THREADS)
		maxthreadspermp = (MAX_CUDA_THREADS/32)*32;
	while (maxthreadspermp > deviceProp.regsPerMultiprocessor * REGISTERS_PER_CUDA_THREAD)
		maxthreadspermp -= 32;
	unsigned minblockspermp = 1;
	while (maxthreadspermp > minblockspermp * deviceProp.maxThreadsPerBlock)
		minblockspermp += 1;
	while (maxthreadspermp * REGISTERS_PER_CUDA_THREAD > minblockspermp * deviceProp.regsPerBlock)
		minblockspermp += 1;

	detail->threadsperblock = ((maxthreadspermp / minblockspermp) / 32) * 32;
	detail->blocks = minblockspermp * deviceProp.multiProcessorCount;
	cout << "Using " << detail->blocks << " blocks with " << detail->threadsperblock << " threads each: total " << detail->blocks * detail->threadsperblock << " threads." << endl;

	CUDA_SAFE_CALL( hipSetDevice(device) );
//	CUDA_SAFE_CALL( hipSetDeviceFlags( hipDeviceScheduleBlockingSync ) );

//	work_buffer_t* trailsout_buf;//[MAX_CUDA_BLOCKS];
//	work_control_t* trailsout_ctl;//[MAX_CUDA_BLOCKS];
//	collisions_buffer_t* collisionsin_buf;//[MAX_CUDA_BLOCKS];
//	collisions_control_t* collisionsin_ctl;//[MAX_CUDA_BLOCKS];
//	collisions_buffer_t* collisionsout_buf;//[MAX_CUDA_BLOCKS];
//	collisions_control_t* collisionsout_ctl;//[MAX_CUDA_BLOCKS];

	CUDA_SAFE_CALL( hipHostMalloc( (void**)(&(detail->trailsout_buf)), detail->blocks * sizeof(work_buffer_t) ) );
	CUDA_SAFE_CALL( hipHostMalloc( (void**)(&(detail->trailsout_ctl)), detail->blocks * sizeof(work_control_t) ) );
	CUDA_SAFE_CALL( hipHostMalloc( (void**)(&(detail->collisionsin_buf)),  sizeof(collisions_buffer_t) ) );
	CUDA_SAFE_CALL( hipHostMalloc( (void**)(&(detail->collisionsin_ctl)),  sizeof(collisions_control_t) ) );
	CUDA_SAFE_CALL( hipHostMalloc( (void**)(&(detail->collisionsout_buf)), sizeof(collisions_buffer_t) ) );
	CUDA_SAFE_CALL( hipHostMalloc( (void**)(&(detail->collisionsout_ctl)), sizeof(collisions_control_t) ) );

	for (unsigned b = 0; b < detail->blocks; ++b)
		detail->trailsout_buf[b].reset(detail->trailsout_ctl[b]);
	detail->collisionsin_buf->reset(*(detail->collisionsin_ctl));
	detail->collisionsout_buf->reset(*(detail->collisionsout_ctl));

	detail->nrcollisions_on_gpu = 0;

	uint32 pc1[4], pc2[4];
	uint32 a = ihv1b[0], b = ihv1b[1], c = ihv1b[2], d = ihv1b[3];
	MD5_FF ( a, b, c, d, msg1b[ 0],  7, 3614090360); /* 1 */
	MD5_FF ( d, a, b, c, msg1b[ 1], 12, 3905402710); /* 2 */
	MD5_FF ( c, d, a, b, msg1b[ 2], 17,  606105819); /* 3 */
	MD5_FF ( b, c, d, a, msg1b[ 3], 22, 3250441966); /* 4 */
	MD5_FF ( a, b, c, d, msg1b[ 4],  7, 4118548399); /* 5 */
	MD5_FF ( d, a, b, c, msg1b[ 5], 12, 1200080426); /* 6 */
	MD5_FF ( c, d, a, b, msg1b[ 6], 17, 2821735955); /* 7 */
	MD5_FF ( b, c, d, a, msg1b[ 7], 22, 4249261313); /* 8 */
	MD5_FF ( a, b, c, d, msg1b[ 8],  7, 1770035416); /* 9 */
	MD5_FF ( d, a, b, c, msg1b[ 9], 12, 2336552879); /* 10 */
	MD5_FF ( c, d, a, b, msg1b[10], 17, 4294925233); /* 11 */
	MD5_FF ( b, c, d, a, msg1b[11], 22, 2304563134); /* 12 */
	MD5_FF ( a, b, c, d, msg1b[12],  7, 1804603682); /* 13 */
	pc1[0] = a; pc1[1] = b; pc1[2] = c; pc1[3] = d;
	a = ihv2b[0]; b = ihv2b[1]; c = ihv2b[2]; d = ihv2b[3];
	MD5_FF ( a, b, c, d, msg2b[ 0],  7, 3614090360); /* 1 */
	MD5_FF ( d, a, b, c, msg2b[ 1], 12, 3905402710); /* 2 */
	MD5_FF ( c, d, a, b, msg2b[ 2], 17,  606105819); /* 3 */
	MD5_FF ( b, c, d, a, msg2b[ 3], 22, 3250441966); /* 4 */
	MD5_FF ( a, b, c, d, msg2b[ 4],  7, 4118548399); /* 5 */
	MD5_FF ( d, a, b, c, msg2b[ 5], 12, 1200080426); /* 6 */
	MD5_FF ( c, d, a, b, msg2b[ 6], 17, 2821735955); /* 7 */
	MD5_FF ( b, c, d, a, msg2b[ 7], 22, 4249261313); /* 8 */
	MD5_FF ( a, b, c, d, msg2b[ 8],  7, 1770035416); /* 9 */
	MD5_FF ( d, a, b, c, msg2b[ 9], 12, 2336552879); /* 10 */
	MD5_FF ( c, d, a, b, msg2b[10], 17, 4294925233); /* 11 */
	MD5_FF ( b, c, d, a, msg2b[11], 22, 2304563134); /* 12 */
	MD5_FF ( a, b, c, d, msg2b[12],  7, 1804603682); /* 13 */
	pc2[0] = a; pc2[1] = b; pc2[2] = c; pc2[3] = d;

	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(msg1), msg1b, sizeof(msg1)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(msg2), msg2b, sizeof(msg2)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(ihv1), ihv1b, sizeof(ihv1)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(ihv2), ihv2b, sizeof(ihv2)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(ihv2mod), ihv2modb, sizeof(ihv2mod)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(precomp1), pc1, sizeof(pc1)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(precomp2), pc2, sizeof(pc2)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(hybridmask), &hmask, sizeof(hmask)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(distinguishedpointmask), &dpmask, sizeof(dpmask)) );
	CUDA_SAFE_CALL( hipMemcpyToSymbol(HIP_SYMBOL(maximumpathlength), &maxlen, sizeof(maxlen)) );


	cuda_md5_init<<<detail->blocks, detail->threadsperblock>>>();

	return true;
}




template<bool mod = false>
__device__ void cuda_md5_work2(uint64 seed)
{
	halt_flag = 0;
	/********************* GENERATE TRAILS ***********************/
	restore_controls();
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint32 len = gworking_states.get<6>(idx);
	uint32 x = gworking_states.get<3>(idx); //end[0]
	uint32 y = gworking_states.get<4>(idx); //end[1]
	uint32 z = gworking_states.get<5>(idx); //end[2]
	if (len >= maximumpathlength || len == 0) {
		x = uint32(seed>>32) ^ threadIdx.x;
		y = uint32(seed) ^ blockIdx.x;
		z = 0;
		gworking_states.get_ref<0>(idx) = x;
		gworking_states.get_ref<1>(idx) = y;
		gworking_states.get_ref<2>(idx) = z;
		len = 0;
	}

	for (unsigned j = 0; j < (1<<12); ++j)
	{
		{
			uint32* in = msg1;
			uint32 a = precomp1[0], b = precomp1[1], c = precomp1[2], d = precomp1[3];
			if (x > y) {
				in = msg2;
				a = precomp2[0]; b = precomp2[1]; c = precomp2[2]; d = precomp2[3];
			}
			MD5_FF ( d, a, b, c, z, 12, 4254626195); /* 14 */
			MD5_FF ( c, d, a, b, x, 17, 2792965006); /* 15 */
			MD5_FF ( b, c, d, a, y, 22, 1236535329); /* 16 */

			MD5_GG ( a, b, c, d, in[ 1],  5, 4129170786); /* 17 */
			MD5_GG ( d, a, b, c, in[ 6],  9, 3225465664); /* 18 */
			MD5_GG ( c, d, a, b, in[11], 14,  643717713); /* 19 */
			MD5_GG ( b, c, d, a, in[ 0], 20, 3921069994); /* 20 */
			MD5_GG ( a, b, c, d, in[ 5],  5, 3593408605); /* 21 */
			MD5_GG ( d, a, b, c, in[10],  9,   38016083); /* 22 */
			MD5_GG ( c, d, a, b, y, 14, 3634488961); /* 23 */
			MD5_GG ( b, c, d, a, in[ 4], 20, 3889429448); /* 24 */
			MD5_GG ( a, b, c, d, in[ 9],  5,  568446438); /* 25 */
			MD5_GG ( d, a, b, c, x,  9, 3275163606); /* 26 */
			MD5_GG ( c, d, a, b, in[ 3], 14, 4107603335); /* 27 */
			MD5_GG ( b, c, d, a, in[ 8], 20, 1163531501); /* 28 */
			MD5_GG ( a, b, c, d, z,  5, 2850285829); /* 29 */
			MD5_GG ( d, a, b, c, in[ 2],  9, 4243563512); /* 30 */
			MD5_GG ( c, d, a, b, in[ 7], 14, 1735328473); /* 31 */
			MD5_GG ( b, c, d, a, in[12], 20, 2368359562); /* 32 */

			MD5_HH ( a, b, c, d, in[ 5],  4, 4294588738); /* 33 */
			MD5_HH ( d, a, b, c, in[ 8], 11, 2272392833); /* 34 */
			MD5_HH ( c, d, a, b, in[11], 16, 1839030562); /* 35 */
			MD5_HH ( b, c, d, a, x, 23, 4259657740); /* 36 */
			MD5_HH ( a, b, c, d, in[ 1],  4, 2763975236); /* 37 */
			MD5_HH ( d, a, b, c, in[ 4], 11, 1272893353); /* 38 */
			MD5_HH ( c, d, a, b, in[ 7], 16, 4139469664); /* 39 */
			MD5_HH ( b, c, d, a, in[10], 23, 3200236656); /* 40 */
			MD5_HH ( a, b, c, d, z,  4,  681279174); /* 41 */
			MD5_HH ( d, a, b, c, in[ 0], 11, 3936430074); /* 42 */
			MD5_HH ( c, d, a, b, in[ 3], 16, 3572445317); /* 43 */
			MD5_HH ( b, c, d, a, in[ 6], 23,   76029189); /* 44 */
			MD5_HH ( a, b, c, d, in[ 9],  4, 3654602809); /* 45 */
			MD5_HH ( d, a, b, c, in[12], 11, 3873151461); /* 46 */
			MD5_HH ( c, d, a, b, y, 16,  530742520); /* 47 */
			MD5_HH ( b, c, d, a, in[ 2], 23, 3299628645); /* 48 */

			MD5_II ( a, b, c, d, in[ 0],  6, 4096336452); /* 49 */
			MD5_II ( d, a, b, c, in[ 7], 10, 1126891415); /* 50 */
			MD5_II ( c, d, a, b, x, 15, 2878612391); /* 51 */
			MD5_II ( b, c, d, a, in[ 5], 21, 4237533241); /* 52 */
			MD5_II ( a, b, c, d, in[12],  6, 1700485571); /* 53 */
			MD5_II ( d, a, b, c, in[ 3], 10, 2399980690); /* 54 */
			MD5_II ( c, d, a, b, in[10], 15, 4293915773); /* 55 */
			MD5_II ( b, c, d, a, in[ 1], 21, 2240044497); /* 56 */
			MD5_II ( a, b, c, d, in[ 8],  6, 1873313359); /* 57 */
			MD5_II ( d, a, b, c, y, 10, 4264355552); /* 58 */
			MD5_II ( c, d, a, b, in[ 6], 15, 2734768916); /* 59 */
			MD5_II ( b, c, d, a, z, 21, 1309151649); /* 60 */
			MD5_II ( a, b, c, d, in[ 4],  6, 4149444226); /* 61 */
			MD5_II ( d, a, b, c, in[11], 10, 3174756917); /* 62 */
			MD5_II ( c, d, a, b, in[ 2], 15,  718787259); /* 63 */
			MD5_II ( b, c, d, a, in[ 9], 21, 3951481745); /* 64 */

			if (mod)
			{
				if (x <= y) {
					x = a + ihv1[0];
					y = d + ihv1[3];
					z = (c + ihv1[2]) & hybridmask;
				} else {
					x = a + ihv2mod[0];
					y = d + ihv2mod[3];
					z = (c + ihv2mod[2]) & hybridmask;
				}
			}
			else
			{
				if (x <= y) {
					a += ihv1[0];
					b += ihv1[1];
					c += ihv1[2];
					d += ihv1[3];
				} else {
					a += ihv2mod[0];
					b += ihv2mod[1];
					c += ihv2mod[2];
					d += ihv2mod[3];
				}
				x = a;
				y = d - c;
				z = (d - b) & hybridmask;
			}
			++len;
		}

		{
			// conditionally write
			bool done = (0 == (x & distinguishedpointmask));
			gtrailsout_buf[blockIdx.x].write(gtrailsout_ctlblock, done,
				gworking_states.get_ref<0>(idx),
				gworking_states.get_ref<1>(idx),
				gworking_states.get_ref<2>(idx),
				x, y, z, len
				);
			if (done)
			{
				x = uint32(seed>>32) ^ (threadIdx.x<<16) + len;
				y = uint32(seed) ^ blockIdx.x;
				z = 0;
				len = 0;
				gworking_states.get_ref<0>(idx) = x;
				gworking_states.get_ref<1>(idx) = y;
				gworking_states.get_ref<2>(idx) = z;
			}
		}
//		__syncthreads();
	}

	gworking_states.get_ref<3>(idx) = x;
	gworking_states.get_ref<4>(idx) = y;
	gworking_states.get_ref<5>(idx) = z;
	gworking_states.get_ref<6>(idx) = len;
	backup_controls();
	halt_flag = 1;
}

template<bool mod = false>
__global__ void cuda_md5_work(uint64 seed)
{
	cuda_md5_work2<mod>(seed);
}


template<bool mod = false>
__global__ void cuda_md5_collisions(uint64 seed)
{
	halt_flag = 0;
	/********** PROCESS COLLIDING TRAILS INTO COLLISIONS ***************/
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;
	uint32 bad = gcollision_states.get<14>(idx);
	uint32 len = gcollision_states.get<6>(idx);
	uint32 len2 = gcollision_states.get<7+6>(idx);
	// if collision state is empty then go read a collision
	if (len == 0 || len2 == 0)
		bad = 1;
	uint32 readidx = gcollisionsin_buf.getreadidx(gcollisionsin_ctl,bad);
	if (bad && readidx < 0xEEEEEEEE)
	{
		len  = gcollisionsin_buf.get<6>(readidx);
		len2 = gcollisionsin_buf.get<7+6>(readidx);
		gcollision_states.get_ref<0>(idx) = gcollisionsin_buf.get<0>(readidx); //start[0]
		gcollision_states.get_ref<1>(idx) = gcollisionsin_buf.get<1>(readidx); //start[1]
		gcollision_states.get_ref<2>(idx) = gcollisionsin_buf.get<2>(readidx); //start[2]
		gcollision_states.get_ref<3>(idx) = gcollisionsin_buf.get<0>(readidx); //start[0]
		gcollision_states.get_ref<4>(idx) = gcollisionsin_buf.get<1>(readidx); //start[1]
		gcollision_states.get_ref<5>(idx) = gcollisionsin_buf.get<2>(readidx); //start[2]
		gcollision_states.get_ref<6>(idx) = gcollisionsin_buf.get<6>(readidx); //len
		gcollision_states.get_ref<7+0>(idx) = gcollisionsin_buf.get<7+0>(readidx);
		gcollision_states.get_ref<7+1>(idx) = gcollisionsin_buf.get<7+1>(readidx);
		gcollision_states.get_ref<7+2>(idx) = gcollisionsin_buf.get<7+2>(readidx);
		gcollision_states.get_ref<7+3>(idx) = gcollisionsin_buf.get<7+0>(readidx);
		gcollision_states.get_ref<7+4>(idx) = gcollisionsin_buf.get<7+1>(readidx);
		gcollision_states.get_ref<7+5>(idx) = gcollisionsin_buf.get<7+2>(readidx);
		gcollision_states.get_ref<7+6>(idx) = gcollisionsin_buf.get<7+6>(readidx);
		gcollision_states.get_ref<14>(idx) = bad = 0;
	}
	if (__all_sync(WARP_FULL_MASK,bad))
	{
//		cuda_md5_work2<mod>(seed);
		return;
	}
	for (unsigned j = 0; j < (1<<12); ++j)
	{
		// always process the longest
		uint32 x, y, z;
		if (len >= len2)
		{
			// process trail1
			// load start+1, write to start
			x = gcollision_states.get<3>(idx);
			y = gcollision_states.get<4>(idx);
			z = gcollision_states.get<5>(idx);
			gcollision_states.get_ref<0>(idx) = x;
			gcollision_states.get_ref<1>(idx) = y;
			gcollision_states.get_ref<2>(idx) = z;
		}
		else
		{
			// process trail2
			// load start+1, write to start
			x = gcollision_states.get<7+3>(idx);
			y = gcollision_states.get<7+4>(idx);
			z = gcollision_states.get<7+5>(idx);
			gcollision_states.get_ref<7+0>(idx) = x;
			gcollision_states.get_ref<7+1>(idx) = y;
			gcollision_states.get_ref<7+2>(idx) = z;
		}

		{
			uint32* in = msg1;
			uint32 a = precomp1[0], b = precomp1[1], c = precomp1[2], d = precomp1[3];
			if (x > y) {
				in = msg2;
				a = precomp2[0]; b = precomp2[1]; c = precomp2[2]; d = precomp2[3];
			}
			MD5_FF ( d, a, b, c, z, 12, 4254626195); /* 14 */
			MD5_FF ( c, d, a, b, x, 17, 2792965006); /* 15 */
			MD5_FF ( b, c, d, a, y, 22, 1236535329); /* 16 */

			MD5_GG ( a, b, c, d, in[ 1],  5, 4129170786); /* 17 */
			MD5_GG ( d, a, b, c, in[ 6],  9, 3225465664); /* 18 */
			MD5_GG ( c, d, a, b, in[11], 14,  643717713); /* 19 */
			MD5_GG ( b, c, d, a, in[ 0], 20, 3921069994); /* 20 */
			MD5_GG ( a, b, c, d, in[ 5],  5, 3593408605); /* 21 */
			MD5_GG ( d, a, b, c, in[10],  9,   38016083); /* 22 */
			MD5_GG ( c, d, a, b, y, 14, 3634488961); /* 23 */
			MD5_GG ( b, c, d, a, in[ 4], 20, 3889429448); /* 24 */
			MD5_GG ( a, b, c, d, in[ 9],  5,  568446438); /* 25 */
			MD5_GG ( d, a, b, c, x,  9, 3275163606); /* 26 */
			MD5_GG ( c, d, a, b, in[ 3], 14, 4107603335); /* 27 */
			MD5_GG ( b, c, d, a, in[ 8], 20, 1163531501); /* 28 */
			MD5_GG ( a, b, c, d, z,  5, 2850285829); /* 29 */
			MD5_GG ( d, a, b, c, in[ 2],  9, 4243563512); /* 30 */
			MD5_GG ( c, d, a, b, in[ 7], 14, 1735328473); /* 31 */
			MD5_GG ( b, c, d, a, in[12], 20, 2368359562); /* 32 */

			MD5_HH ( a, b, c, d, in[ 5],  4, 4294588738); /* 33 */
			MD5_HH ( d, a, b, c, in[ 8], 11, 2272392833); /* 34 */
			MD5_HH ( c, d, a, b, in[11], 16, 1839030562); /* 35 */
			MD5_HH ( b, c, d, a, x, 23, 4259657740); /* 36 */
			MD5_HH ( a, b, c, d, in[ 1],  4, 2763975236); /* 37 */
			MD5_HH ( d, a, b, c, in[ 4], 11, 1272893353); /* 38 */
			MD5_HH ( c, d, a, b, in[ 7], 16, 4139469664); /* 39 */
			MD5_HH ( b, c, d, a, in[10], 23, 3200236656); /* 40 */
			MD5_HH ( a, b, c, d, z,  4,  681279174); /* 41 */
			MD5_HH ( d, a, b, c, in[ 0], 11, 3936430074); /* 42 */
			MD5_HH ( c, d, a, b, in[ 3], 16, 3572445317); /* 43 */
			MD5_HH ( b, c, d, a, in[ 6], 23,   76029189); /* 44 */
			MD5_HH ( a, b, c, d, in[ 9],  4, 3654602809); /* 45 */
			MD5_HH ( d, a, b, c, in[12], 11, 3873151461); /* 46 */
			MD5_HH ( c, d, a, b, y, 16,  530742520); /* 47 */
			MD5_HH ( b, c, d, a, in[ 2], 23, 3299628645); /* 48 */

			MD5_II ( a, b, c, d, in[ 0],  6, 4096336452); /* 49 */
			MD5_II ( d, a, b, c, in[ 7], 10, 1126891415); /* 50 */
			MD5_II ( c, d, a, b, x, 15, 2878612391); /* 51 */
			MD5_II ( b, c, d, a, in[ 5], 21, 4237533241); /* 52 */
			MD5_II ( a, b, c, d, in[12],  6, 1700485571); /* 53 */
			MD5_II ( d, a, b, c, in[ 3], 10, 2399980690); /* 54 */
			MD5_II ( c, d, a, b, in[10], 15, 4293915773); /* 55 */
			MD5_II ( b, c, d, a, in[ 1], 21, 2240044497); /* 56 */
			MD5_II ( a, b, c, d, in[ 8],  6, 1873313359); /* 57 */
			MD5_II ( d, a, b, c, y, 10, 4264355552); /* 58 */
			MD5_II ( c, d, a, b, in[ 6], 15, 2734768916); /* 59 */
			MD5_II ( b, c, d, a, z, 21, 1309151649); /* 60 */
			MD5_II ( a, b, c, d, in[ 4],  6, 4149444226); /* 61 */
			MD5_II ( d, a, b, c, in[11], 10, 3174756917); /* 62 */
			MD5_II ( c, d, a, b, in[ 2], 15,  718787259); /* 63 */
			MD5_II ( b, c, d, a, in[ 9], 21, 3951481745); /* 64 */

			if (mod)
			{
				if (x <= y) {
					x = a + ihv1[0];
					y = d + ihv1[3];
					z = (c + ihv1[2]) & hybridmask;
				} else {
					x = a + ihv2mod[0];
					y = d + ihv2mod[3];
					z = (c + ihv2mod[2]) & hybridmask;
				}
			}
			else
			{
				if (x <= y) {
					a += ihv1[0];
					b += ihv1[1];
					c += ihv1[2];
					d += ihv1[3];
				} else {
					a += ihv2mod[0];
					b += ihv2mod[1];
					c += ihv2mod[2];
					d += ihv2mod[3];
				}
				x = a;
				y = d - c;
				z = (d - b) & hybridmask;
			}
		}

		if (len >= len2)
		{
			// processed trail1
			// write to end
			gcollision_states.get_ref<3>(idx) = x;
			gcollision_states.get_ref<4>(idx) = y;
			gcollision_states.get_ref<5>(idx) = z;
			if (len > 0)
				--len;
		}
		else
		{
			// processed trail2
			// write to end
			gcollision_states.get_ref<7+3>(idx) = x;
			gcollision_states.get_ref<7+4>(idx) = y;
			gcollision_states.get_ref<7+5>(idx) = z;
			if (len2 > 0)
				--len2;
		}

		bool done = (bad == 0)
			&& (len == 0 || len2 == 0 || 
				(
				  (gcollision_states.get<3>(idx) == gcollision_states.get<7+3>(idx))
				  && (gcollision_states.get<4>(idx) == gcollision_states.get<7+4>(idx))
				  && (gcollision_states.get<5>(idx) == gcollision_states.get<7+5>(idx))
				));

		{
			if (done)
			{
				if (len > 0) len = 1;
				if (len2 > 0) len2 = 1;
				bad = 1;
			}
			// conditionally write result and load a new one
			gcollisionsout_buf.write(gcollisionsout_ctl, done,
				gcollision_states.get<0>(idx),
				gcollision_states.get<1>(idx),
				gcollision_states.get<2>(idx),
				gcollision_states.get<3>(idx),
				gcollision_states.get<4>(idx),
				gcollision_states.get<5>(idx),
				len,
				gcollision_states.get<7+0>(idx),
				gcollision_states.get<7+1>(idx),
				gcollision_states.get<7+2>(idx),
				gcollision_states.get<7+3>(idx),
				gcollision_states.get<7+4>(idx),
				gcollision_states.get<7+5>(idx),
				len2);
		}

		if (4 <= __popc(__ballot_sync(WARP_FULL_MASK,bad)))
		{
			uint32 readidx = gcollisionsin_buf.getreadidx(gcollisionsin_ctl, bad);
			if (bad && readidx < 0xEEEEEEEE)
			{
				len  = gcollisionsin_buf.get<6>(readidx);
				len2 = gcollisionsin_buf.get<7+6>(readidx);
				gcollision_states.get_ref<0>(idx) = gcollisionsin_buf.get<0>(readidx);
				gcollision_states.get_ref<1>(idx) = gcollisionsin_buf.get<1>(readidx);
				gcollision_states.get_ref<2>(idx) = gcollisionsin_buf.get<2>(readidx);
				gcollision_states.get_ref<3>(idx) = gcollisionsin_buf.get<0>(readidx);
				gcollision_states.get_ref<4>(idx) = gcollisionsin_buf.get<1>(readidx);
				gcollision_states.get_ref<5>(idx) = gcollisionsin_buf.get<2>(readidx);
				gcollision_states.get_ref<6>(idx) = gcollisionsin_buf.get<6>(readidx);
				gcollision_states.get_ref<7+0>(idx) = gcollisionsin_buf.get<7+0>(readidx);
				gcollision_states.get_ref<7+1>(idx) = gcollisionsin_buf.get<7+1>(readidx);
				gcollision_states.get_ref<7+2>(idx) = gcollisionsin_buf.get<7+2>(readidx);
				gcollision_states.get_ref<7+3>(idx) = gcollisionsin_buf.get<7+0>(readidx);
				gcollision_states.get_ref<7+4>(idx) = gcollisionsin_buf.get<7+1>(readidx);
				gcollision_states.get_ref<7+5>(idx) = gcollisionsin_buf.get<7+2>(readidx);
				gcollision_states.get_ref<7+6>(idx) = gcollisionsin_buf.get<7+6>(readidx);
				gcollision_states.get_ref<14>(idx) = bad = 0;
			}
		}
		if (__all_sync(WARP_FULL_MASK,bad))
			break;
		if (__shfl_sync(WARP_FULL_MASK,halt_flag,0)) // read global halt flag together and halt if set
			break;
//		__syncthreads();
	}
	gcollision_states.get_ref<6>(idx) = len;
	gcollision_states.get_ref<7+6>(idx) = len2;
	gcollision_states.get_ref<14>(idx) = bad;
}



void cuda_device::cuda_fill_trail_buffer(uint32 id, uint64 seed,
							vector<trail_type>& buf,
							vector< pair<trail_type,trail_type> >& collisions, bool mod)
{
//	CUDA_SAFE_CALL( hipHostMalloc( (void**)(&(detail->trailsout_buf)), detail->blocks * sizeof(work_buffer_t) ) );
//	CUDA_SAFE_CALL( hipHostMalloc( (void**)(&(detail->trailsout_ctl)), detail->blocks * sizeof(work_control_t) ) );

	// move all collisions into buffer
	for (auto& c : collisions)
		detail->collisions.emplace_back(c);
	collisions.clear();

	// if collisions buffer is big enough then actually launch it
	uint32 collisionblocks = 0;
	if (detail->collisions.size())
	{
		size_t oldsize = detail->collisions.size();
		// store input collisions to GPU by writing to host buffer
		// and sending it to GPU, we only move the control back and forth
		uint32 count = detail->collisions.size();
		// don't overwrite collision data still in the buffer
		if (count >= detail->collisionsin_ctl->free_count())
			count = detail->collisionsin_ctl->free_count();
		if (count > 0)
			count -= 1;
		for (std::size_t i = 0; i < count; ++i)
		{
				detail->collisionsin_buf->write(*(detail->collisionsin_ctl), true,
					detail->collisions[i].first.start[0], detail->collisions[i].first.start[1], detail->collisions[i].first.start[2],
					detail->collisions[i].first.end[0], detail->collisions[i].first.end[1], detail->collisions[i].first.end[2],
					detail->collisions[i].first.len,
					detail->collisions[i].second.start[0], detail->collisions[i].second.start[1], detail->collisions[i].second.start[2],
					detail->collisions[i].second.end[0], detail->collisions[i].second.end[1], detail->collisions[i].second.end[2],
					detail->collisions[i].second.len);
		}
		detail->collisions.erase(detail->collisions.begin(), detail->collisions.begin() + count);
		detail->nrcollisions_on_gpu += count;

		// determine how many cuda blocks to start for collision
		collisionblocks = (detail->nrcollisions_on_gpu / detail->threadsperblock)/2;
		if (collisionblocks > detail->blocks)
			collisionblocks = detail->blocks;
		// only copy data to GPU when we're actually going to run GPU code
		if (collisionblocks > 0)
		{
			// send control and buffer structures to GPU
			hipMemcpyToSymbol(HIP_SYMBOL(gcollisionsin_ctl), detail->collisionsin_ctl, sizeof(collisions_control_t));
			hipMemcpyToSymbol(HIP_SYMBOL(gcollisionsin_buf), detail->collisionsin_buf, sizeof(collisions_buffer_t));
		}
		if (0) std::cout << "C: " << oldsize << " " << detail->collisions.size() 
			<< " " << detail->collisionsin_ctl->used_count()
			<< " " << detail->collisionsin_ctl->free_count()
			<< " " << collisionblocks << " " << detail->blocks
			<< std::endl;
		
	}

	// send control structures to GPU
	hipMemcpyToSymbol(HIP_SYMBOL(gtrailsout_ctl), detail->trailsout_ctl, detail->blocks * sizeof(work_control_t));
	// retrieve store buffers from GPU
	hipMemcpyToSymbol(HIP_SYMBOL(gtrailsout_buf), detail->trailsout_buf, detail->blocks * sizeof(work_buffer_t));

	// run GPU code
	if (mod)
	{
		cuda_md5_work<true><<<detail->blocks - collisionblocks, detail->threadsperblock>>>(seed);
		cuda_md5_collisions<true><<< collisionblocks, detail->threadsperblock>>>(seed);
	}
	else
	{
		cuda_md5_work<false><<<detail->blocks - collisionblocks, detail->threadsperblock>>>(seed);
		cuda_md5_collisions<false><<< collisionblocks, detail->threadsperblock>>>(seed);
	}

	// retrieve store buffers from GPU
	hipMemcpyFromSymbol(detail->trailsout_buf, HIP_SYMBOL(gtrailsout_buf), detail->blocks * sizeof(work_buffer_t));
	// retrieve control structures from GPU
	hipMemcpyFromSymbol(detail->trailsout_ctl, HIP_SYMBOL(gtrailsout_ctl), detail->blocks * sizeof(work_control_t));

/*	std::cout << detail->trailsout_ctl[0].write_idx << " " <<
		detail->trailsout_ctl[0].read_idx << std::endl;
*/
	// if we started a collision processing cuda job then process its output
	if (collisionblocks > 0)
	{
		hipMemcpyFromSymbol(detail->collisionsout_buf, HIP_SYMBOL(gcollisionsout_buf), sizeof(collisions_buffer_t));
		hipMemcpyFromSymbol(detail->collisionsin_ctl, HIP_SYMBOL(gcollisionsin_ctl), sizeof(collisions_control_t));
		hipMemcpyFromSymbol(detail->collisionsout_ctl, HIP_SYMBOL(gcollisionsout_ctl), sizeof(collisions_control_t));
		uint32 readidx;
		while ((readidx=detail->collisionsout_buf->getreadidx(*(detail->collisionsout_ctl)))
			< 0xEEEEEEEE)
		{
			--detail->nrcollisions_on_gpu;
			collisions.emplace_back();
			trail_type& first = collisions.back().first;
			trail_type& second = collisions.back().second;
			first.start[0]  = detail->collisionsout_buf->get<0>(readidx);
			first.start[1]  = detail->collisionsout_buf->get<1>(readidx);
			first.start[2]  = detail->collisionsout_buf->get<2>(readidx);
			first.end[0]    = detail->collisionsout_buf->get<3>(readidx);
			first.end[1]    = detail->collisionsout_buf->get<4>(readidx);
			first.end[2]    = detail->collisionsout_buf->get<5>(readidx);
			first.len       = detail->collisionsout_buf->get<6>(readidx);
			second.start[0] = detail->collisionsout_buf->get<7+0>(readidx);
			second.start[1] = detail->collisionsout_buf->get<7+1>(readidx);
			second.start[2] = detail->collisionsout_buf->get<7+2>(readidx);
			second.end[0]   = detail->collisionsout_buf->get<7+3>(readidx);
			second.end[1]   = detail->collisionsout_buf->get<7+4>(readidx);
			second.end[2]   = detail->collisionsout_buf->get<7+5>(readidx);
			second.len      = detail->collisionsout_buf->get<7+6>(readidx);
		}
		hipMemcpyToSymbol(HIP_SYMBOL(gcollisionsout_ctl), detail->collisionsout_ctl, sizeof(collisions_control_t));
	}

	// process and return results
	buf.clear();
	for (unsigned b = 0; b < detail->blocks; ++b)
	{
		uint32 readidx;
		trail_type trail;
		while ((readidx=detail->trailsout_buf[b].getreadidx(detail->trailsout_ctl[b])) != 0xFFFFFFFF)
		{
			trail.start[0] = detail->trailsout_buf[b].get<0>(readidx);
			trail.start[1] = detail->trailsout_buf[b].get<1>(readidx);
			trail.start[2] = detail->trailsout_buf[b].get<2>(readidx);
			trail.end[0]   = detail->trailsout_buf[b].get<3>(readidx);
			trail.end[1]   = detail->trailsout_buf[b].get<4>(readidx);
			trail.end[2]   = detail->trailsout_buf[b].get<5>(readidx);
			trail.len      = detail->trailsout_buf[b].get<6>(readidx);
			buf.push_back(trail);
		}
	}
//	std::cout << "B " << buf.size() << std::endl;
}










#ifdef _WIN32
#include <windows.h>
#else
#include <sys/time.h>
#endif

	class timer_detail;
	class timer {
	public:
		timer(bool direct_start = false);
		~timer();
		void start();
		void stop();
		double time() const;// get time between start and stop (or now if still running) in seconds
		bool isrunning() const { return running; } // check if timer is running

	private:
		timer_detail* detail;
		bool running;
	};
	class timer_detail {
	public:
#ifdef _WIN32
		LARGE_INTEGER tstart, tend;
		double freq;
#else
		struct timeval tstart, tend;
		struct timezone tz;
#endif
	};

	timer::~timer()
	{
		delete detail;
	}

	timer::timer(bool direct_start): running(false)
	{
		detail = new timer_detail;
#ifdef _WIN32
		LARGE_INTEGER tmp_freq;
		QueryPerformanceFrequency(&tmp_freq);
		detail->freq = double(tmp_freq.QuadPart);
#endif
		if (direct_start)
			start();
	}

#ifdef _WIN32

	void timer::start()
	{
		running = true;
		QueryPerformanceCounter(&detail->tstart);
	}

	void timer::stop()
	{
		QueryPerformanceCounter(&detail->tend);
		running = false;
	}

	double timer::time() const
	{
		if (running)
		{
			LARGE_INTEGER tmp_end;
			QueryPerformanceCounter(&tmp_end);
			return (double(tmp_end.QuadPart) - double(detail->tstart.QuadPart))/detail->freq;
		} else
			return (double(detail->tend.QuadPart) - double(detail->tstart.QuadPart))/detail->freq;
	}

#else

	void timer::start()
	{
		running = true;
		gettimeofday(&detail->tstart, &detail->tz);
	}

	void timer::stop()
	{
		gettimeofday(&detail->tend, &detail->tz);
		running = false;
	}

	double timer::time() const
	{
		double t1 = double(detail->tstart.tv_sec) + (double(detail->tstart.tv_usec)/1e6);
		if (running)
		{
			struct timeval tmp_end;
			gettimeofday(&tmp_end, &detail->tz);
			return double(tmp_end.tv_sec) + (double(tmp_end.tv_usec)/1e6) - t1;
		} else
			return double(detail->tend.tv_sec) + (double(detail->tend.tv_usec)/1e6) - t1;
	}

#endif

void cuda_device::benchmark()
{
/*
	timer sw;
	for (int blocksize = 4; blocksize <= 256; ++blocksize)
	for (int threadsize = 250; threadsize <= 257; ++threadsize)
	{
		sw.start();
		uint64 work = 0;
		while (sw.time() < 10) {
			cuda_md5_work<<<blocksize, threadsize>>>(0);
			hipMemcpyFromSymbol(detail->buffer_host, HIP_SYMBOL(buffer2), sizeof(trail_type)*blocksize*threadsize);
			++work;
		}
		uint64 ow = work;
		work *= 0x400 * blocksize * threadsize;
		cout << blocksize << "x" << threadsize << ":\t" << work << " (" << ow << ")" << endl;
	}
*/
}














int get_num_cuda_devices()
{
	int deviceCount = 0;
	cutilSafeCall(hipGetDeviceCount(&deviceCount));
	return deviceCount;
}

void cuda_device_query()
{
    int deviceCount = 0;
    cutilSafeCall(hipGetDeviceCount(&deviceCount));
    if (deviceCount == 0)
        printf("There is no device supporting CUDA\n");
    int dev;
    for (dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t deviceProp;
        cutilSafeCall(hipGetDeviceProperties(&deviceProp, dev));
        if (dev == 0) {
            if (deviceProp.major == 9999 && deviceProp.minor == 9999)
                printf("There is no device supporting CUDA.\n");
            else if (deviceCount == 1)
                printf("There is 1 device supporting CUDA\n");
            else
                printf("There are %d devices supporting CUDA\n", deviceCount);
        }
        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
        printf("  Major revision number:                         %d\n",
               deviceProp.major);
        printf("  Minor revision number:                         %d\n",
               deviceProp.minor);
        printf("  Total amount of global memory:                 %u bytes\n",
               deviceProp.totalGlobalMem);
    #if CUDART_VERSION >= 2000
        printf("  Number of multiprocessors:                     %d\n",
               deviceProp.multiProcessorCount);
        printf("  Number of cores:                               %d\n",
               8 * deviceProp.multiProcessorCount);
    #endif
        printf("  Total amount of constant memory:               %u bytes\n",
               deviceProp.totalConstMem);
        printf("  Total amount of shared memory per block:       %u bytes\n",
               deviceProp.sharedMemPerBlock);
        printf("  Total number of registers available per block: %d\n",
               deviceProp.regsPerBlock);
        printf("  Warp size:                                     %d\n",
               deviceProp.warpSize);
        printf("  Maximum number of threads per block:           %d\n",
               deviceProp.maxThreadsPerBlock);
        printf("  Maximum sizes of each dimension of a block:    %d x %d x %d\n",
               deviceProp.maxThreadsDim[0],
               deviceProp.maxThreadsDim[1],
               deviceProp.maxThreadsDim[2]);
        printf("  Maximum sizes of each dimension of a grid:     %d x %d x %d\n",
               deviceProp.maxGridSize[0],
               deviceProp.maxGridSize[1],
               deviceProp.maxGridSize[2]);
        printf("  Maximum memory pitch:                          %u bytes\n",
               deviceProp.memPitch);
        printf("  Texture alignment:                             %u bytes\n",
               deviceProp.textureAlignment);
        printf("  Clock rate:                                    %.2f GHz\n",
               deviceProp.clockRate * 1e-6f);
    #if CUDART_VERSION >= 2000
        printf("  Concurrent copy and execution:                 %s\n",
               deviceProp.deviceOverlap ? "Yes" : "No");
    #endif
    }

}
